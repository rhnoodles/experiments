#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <limits.h>
#include <linux/mman.h>
#include <sys/mman.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const double *A, const double *B, double *C,
                          unsigned long long int numElements) {
  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

/**
 * Host main routine
 */
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  //int numElements = INT_MAX;
  unsigned long long int numElements = 1600000000;
  size_t size = numElements * sizeof(double);
  printf("[Vector addition of %llu elements, each vector has size %ld MB]\n", numElements, size/(1024*1024));

  // Allocate the host input vector A
  //double *h_A = (double *)malloc(size);
  //double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
  //double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_2MB, -1, 0);
  //double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_1GB, -1, 0);
#ifdef FOURKB
  double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
  double *h_B = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
  double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
#elif TWOMB
  double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_2MB, -1, 0);
  double *h_B = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_2MB, -1, 0);
  double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_2MB, -1, 0);
#elif ONEGB
  double *h_A = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_1GB, -1, 0);
  double *h_B = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_1GB, -1, 0);
  double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_1GB, -1, 0);
#endif

  if (h_A == MAP_FAILED) {
    perror("mmapA");
  }
  
  // Allocate the host input vector B
  //double *h_B = (double *)malloc(size);
  if (h_B == MAP_FAILED) {
    perror("mmapB");
  }
  
  // Allocate the host output vector C
  //double *h_C = (double *)malloc(size);
  //double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);
  //double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_2MB, -1, 0);
  //double *h_C = (double *) mmap(NULL, size, PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANONYMOUS | MAP_HUGETLB | MAP_HUGE_1GB, -1, 0);
  if (h_C == MAP_FAILED) {
    perror("mmapC");
  }
  
  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (unsigned long long int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (double)RAND_MAX;
    h_B[i] = rand() / (double)RAND_MAX;
  }

  // Allocate the device input vector A
  double *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  double *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  double *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  unsigned long long int threadsPerBlock = 256;
  unsigned long long int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %llu blocks of %llu threads\n", blocksPerGrid,
         threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify that the result vector is correct
  for (unsigned long long int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %llu!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  //free(h_A);
  //free(h_B);
  //free(h_C);
  munmap(h_A, size);
  munmap(h_B, size);
  munmap(h_C, size);
  
  printf("Done\n");
  return 0;
}
